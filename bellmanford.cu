#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include "CycleTimer.h"
#include <hip/hip_runtime.h>

#define INF 1e9

struct Edge{
    int src;
    int dst;
    int weight;
};

__global__ BellmanFordKernel(std::vector<Edge>& edges, int srcNode, int dstNode, int numNodes){

    std::vector<int> dist(numNodes, INF);

    int N = numNodes - 1;
    dist[srcNode] = 0;

    int edge_num = edges.size();

    // while (N--){
    //     for (const auto& edge : edges){
    //         if (dist[edge.src] != INF)
    //             dist[edge.dst] = std::min(dist[edge.src] + edge.weight, dist[edge.dst]);
    //     }
    // }

    omp_set_num_threads(4);

    std::vector<std::vector<int>> local_dist(4, std::vector<int>(dist));

    while (N--) {

        // for(int i = 0; i < numNodes; i++){
        //     std::printf("local_dist[%d] = %d\n", i, local_dist[0][i]);
        // }
        // std::printf("\n");

        #pragma omp parallel
        {
            int id = omp_get_thread_num();            

            #pragma omp for
            for (int i = 0; i < edge_num; i++){
                Edge edge = edges[i];

                if (local_dist[id][edge.src] != INF){
                    local_dist[id][edge.dst] = std::min(local_dist[id][edge.src] + edge.weight, local_dist[id][edge.dst]);
                }      
            }
            
            #pragma omp for
            for (int i = 0; i < numNodes; i++){
                dist[i] = std::min({local_dist[0][i], local_dist[1][i], local_dist[2][i], local_dist[3][i]});
                // std::printf("dist[%d] = %d\n", i, dist[i]);
            }

            for (int i = 0; i < numNodes; i++){
                local_dist[id][i] = dist[i];
            }
            
        }

        
    }

    // #pragma omp parallel for
    for (int i = 0; i < edge_num; i++){
        Edge edge = edges[i];
        if (dist[edge.src] != INF && dist[edge.dst] > dist[edge.src] + edge.weight){
            std::printf("NEGATIVE CYCLE !!\n");
            return -1;
        }
    }

    return dist[dstNode];
    
}

int BellmanFord(Edge* edges, int srcNode, int dstNode, int numNodes, int numEdges){
    Edge* deviceArray;
    hipMalloc(&deviceArray, numEdges * sizeof(struct Edge));

    
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::printf("usage: ./bellmanford_thread file.txt srcNode dstNode\n");
        return 1;
    }
    
    std::ifstream ifs;
    ifs.open(argv[1], std::ifstream::in);
    if (!ifs.good()) {
        std::printf("[Error] Cannot open file %s\n", argv[1]);
        return 1;
    }

    int numNodes, numEdges;
    ifs >> numNodes >> numEdges;
    std::printf("[numNode]: %d [numEdges]: %d\n", numNodes, numEdges);

    Edge* edges = (Edge*)malloc(numEdges * 2 * sizeof(struct Edge));

    int srcNode = std::atoi(argv[2]);
    int dstNode = std::atoi(argv[3]);
    std::printf("[srcNode]: %u [dstNode]: %u\n", srcNode, dstNode);

    if (srcNode >= numNodes || dstNode >= numNodes){
        std::printf("[Error] Given src or dst node is in the graph\n");
        return 1;
    }

    int source, target, weight;
    for (int i = 0; i < numEdges; i++) {
        ifs >> source >> target >> weight;

        // Edge e1 = {source, target, weight};
        // edges.push_back(e1);
        edges[i*2].src = source;
        edges[i*2+1].src = source;

        edges[i*2].dst = target;
        edges[i*2+1].dst = target;

        edges[i*2].weight = weight;
        edges[i*2+1].weight = weight;

        // Edge e2 = {target, source, weight};
        // edges.push_back(e2);
    }

    std::printf("Successfully construct Edge vector\n");
    ifs.close();

    double startTime = CycleTimer::currentSeconds();
    int minDist = BellmanFord(edges, srcNode, dstNode, numNodes, numEdges*2);
    double endTime = CycleTimer::currentSeconds();

    std::printf("[BellmanFord thread]:\t\t[%lf] ms\n", (endTime - startTime) * 1000);
    std::printf("The minimum distance from %d to %d is: %d\n", srcNode, dstNode, minDist);
    
    return 0;
}