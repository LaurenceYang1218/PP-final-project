#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include "CycleTimer.h"
#include <hip/hip_runtime.h>

#define INF 1e9
#define ITER_NUM 10

struct Edge{
    int src;
    int dst;
    int weight;
};

__global__ void BellmanFordKernel(Edge* edges, int* dist, int* parent, int numNodes, int numEdges){

    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId >= numEdges)
        return;

    Edge edge = edges[threadId];
    // printf("threadId: %d\n", threadId);

    if (dist[edge.src] != INF && dist[edge.src] + edge.weight < dist[edge.dst]){
        atomicMin(&dist[edge.dst], dist[edge.src] + edge.weight);

        int oldParent = atomicCAS(&parent[edge.dst], parent[edge.dst], edge.src);
    }
        
}

int BellmanFord(Edge* edges, int srcNode, int dstNode, int numNodes, int numEdges, bool printRoute){
    Edge* deviceEdgeArray;
    hipMalloc(&deviceEdgeArray, numEdges * sizeof(struct Edge));
    hipMemcpy(deviceEdgeArray, edges, numEdges * sizeof(struct Edge), hipMemcpyHostToDevice);
    
    int* dist = (int*)malloc(numNodes * sizeof(int));
    int* parent = (int*)malloc(numNodes * sizeof(int));
    
    for (int i = 0; i < numNodes; i++){
        dist[i] = INF;
        parent[i] = -1;
    }

    dist[srcNode] = 0;

    int* deviceDistArray;
    hipMalloc(&deviceDistArray, numNodes * sizeof(int));
    hipMemcpy(deviceDistArray, dist, numNodes * sizeof(int), hipMemcpyHostToDevice);

    int* deviceParentArray;
    hipMalloc(&deviceParentArray, numNodes * sizeof(int));
    hipMemcpy(deviceParentArray, parent, numNodes * sizeof(int), hipMemcpyHostToDevice);
    
    int N = numNodes - 1;
    int threadPerBlock = 64;
    int numBlocks = (numEdges + threadPerBlock - 1) / threadPerBlock;

    while(N--){
        BellmanFordKernel<<<numBlocks, threadPerBlock>>>(deviceEdgeArray, deviceDistArray, deviceParentArray, numNodes, numEdges);
    }

    hipMemcpy(dist, deviceDistArray, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < numEdges; i++){
        Edge edge = edges[i];
        if (dist[edge.src] != INF && dist[edge.dst] > dist[edge.src] + edge.weight){
            std::printf("NEGATIVE CYCLE !!\n");
            return -1;
        }
    }

    hipFree(deviceEdgeArray);
    hipFree(deviceDistArray);
    
    int min_dist = dist[dstNode];
    free(dist);

    if (printRoute){
        int* route = (int*)malloc(numNodes * sizeof(int));
        int numRouteNodes = 1;
        route[0] = dstNode;

        int curNode = dstNode;
        
        while(curNode != srcNode){
            route[numRouteNodes] = parent[curNode];
            curNode = parent[curNode];
            numRouteNodes++;
        }

        std::printf("The shortest path is:\n");
    
        for (int i = numRouteNodes-1; i >= 1; i--){
            std::printf("%d -> ", route[i]);
        }

        std::printf("%d\n", route[0]);

        free(route);
    }

    return min_dist;
    
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::printf("usage: ./bellmanford_cuda file.txt srcNode dstNode\n");
        return 1;
    }
    
    std::ifstream ifs;
    ifs.open(argv[1], std::ifstream::in);
    if (!ifs.good()) {
        std::printf("[Error] Cannot open file %s\n", argv[1]);
        return 1;
    }

    int numNodes, numEdges;
    ifs >> numNodes >> numEdges;
    std::printf("[numNodes]: %d [numEdges]: %d\n", numNodes, numEdges);

    Edge* edges = (Edge*)malloc(numEdges * 2 * sizeof(struct Edge));

    int srcNode = std::atoi(argv[2]);
    int dstNode = std::atoi(argv[3]);
    std::printf("[srcNode]: %u [dstNode]: %u\n", srcNode, dstNode);

    if (srcNode >= numNodes || dstNode >= numNodes){
        std::printf("[Error] Given src or dst node is in the graph\n");
        return 1;
    }

    int source, target, weight;
    for (int i = 0; i < numEdges; i++) {
        ifs >> source >> target >> weight;

        // Edge e1 = {source, target, weight};
        // edges.push_back(e1);
        edges[i*2].src = source;
        edges[i*2+1].src = target;

        edges[i*2].dst = target;
        edges[i*2+1].dst = source;

        edges[i*2].weight = weight;
        edges[i*2+1].weight = weight;

        // Edge e2 = {target, source, weight};
        // edges.push_back(e2);
    }

    std::printf("Successfully construct Edge vector\n");
    ifs.close();

    double avgTime = 0.0;
    int minDist = 0;
    bool printRoute = false;

    for (int i = 0; i < ITER_NUM; i++){
        double startTime = CycleTimer::currentSeconds();
        minDist = BellmanFord(edges, srcNode, dstNode, numNodes, numEdges*2, printRoute);
        double endTime = CycleTimer::currentSeconds();

        avgTime += endTime - startTime;
    }

    avgTime /= ITER_NUM;

    printRoute = true;
    std::printf("[BellmanFord cuda]:\t\t[%lf] ms\n", avgTime * 1000);
    minDist = BellmanFord(edges, srcNode, dstNode, numNodes, numEdges*2, printRoute);
    std::printf("The minimum distance from %d to %d is: %d\n\n", srcNode, dstNode, minDist);
    
    return 0;
}