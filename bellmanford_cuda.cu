#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include "CycleTimer.h"
#include <hip/hip_runtime.h>

#define INF 1e9
#define ITER_NUM 10

struct Edge{
    int src;
    int dst;
    int weight;
};

__global__ void BellmanFordKernel(Edge* edges, int* dist, int numNodes, int numEdges){

    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId >= numEdges)
        return;

    Edge edge = edges[threadId];

    if (dist[edge.src] != INF/*  && dist[edge.src] + edge.weight < dist[edge.dst] */)
        atomicMin(&dist[edge.dst], dist[edge.src] + edge.weight);
        
}

int BellmanFord(Edge* edges, int srcNode, int dstNode, int numNodes, int numEdges){
    Edge* deviceEdgeArray;
    hipMalloc(&deviceEdgeArray, numEdges * sizeof(struct Edge));
    hipMemcpy(deviceEdgeArray, edges, numEdges * sizeof(struct Edge), hipMemcpyHostToDevice);
    
    int* dist = (int*)malloc(numNodes * sizeof(int));
    
    for (int i = 0; i < numNodes; i++){
        dist[i] = INF;
    }

    dist[srcNode] = 0;

    int* deviceDistArray;
    hipMalloc(&deviceDistArray, numNodes * sizeof(int));
    hipMemcpy(deviceDistArray, dist, numNodes * sizeof(int), hipMemcpyHostToDevice);
    
    int N = numNodes - 1;
    int threadPerBlock = 64;
    int numBlocks = (numEdges + threadPerBlock - 1) / threadPerBlock;

    while(N--){
        BellmanFordKernel<<<numBlocks, threadPerBlock>>>(deviceEdgeArray, deviceDistArray, numNodes, numEdges);
    }

    hipMemcpy(dist, deviceDistArray, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < numEdges; i++){
        Edge edge = edges[i];
        if (dist[edge.src] != INF && dist[edge.dst] > dist[edge.src] + edge.weight){
            std::printf("NEGATIVE CYCLE !!\n");
            return -1;
        }
    }

    hipFree(deviceEdgeArray);
    hipFree(deviceDistArray);
    
    int min_dist = dist[dstNode];
    free(dist);

    return min_dist;
    
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::printf("usage: ./bellmanford_cuda file.txt srcNode dstNode\n");
        return 1;
    }
    
    std::ifstream ifs;
    ifs.open(argv[1], std::ifstream::in);
    if (!ifs.good()) {
        std::printf("[Error] Cannot open file %s\n", argv[1]);
        return 1;
    }

    int numNodes, numEdges;
    ifs >> numNodes >> numEdges;
    std::printf("[numNodes]: %d [numEdges]: %d\n", numNodes, numEdges);

    Edge* edges = (Edge*)malloc(numEdges * 2 * sizeof(struct Edge));

    int srcNode = std::atoi(argv[2]);
    int dstNode = std::atoi(argv[3]);
    std::printf("[srcNode]: %u [dstNode]: %u\n", srcNode, dstNode);

    if (srcNode >= numNodes || dstNode >= numNodes){
        std::printf("[Error] Given src or dst node is in the graph\n");
        return 1;
    }

    int source, target, weight;
    for (int i = 0; i < numEdges; i++) {
        ifs >> source >> target >> weight;

        // Edge e1 = {source, target, weight};
        // edges.push_back(e1);
        edges[i*2].src = source;
        edges[i*2+1].src = target;

        edges[i*2].dst = target;
        edges[i*2+1].dst = source;

        edges[i*2].weight = weight;
        edges[i*2+1].weight = weight;

        // Edge e2 = {target, source, weight};
        // edges.push_back(e2);
    }

    std::printf("Successfully construct Edge vector\n");
    ifs.close();

    double avgTime = 0.0;
    int minDist = 0;

    for (int i = 0; i < ITER_NUM; i++){
        double startTime = CycleTimer::currentSeconds();
        minDist = BellmanFord(edges, srcNode, dstNode, numNodes, numEdges*2);
        double endTime = CycleTimer::currentSeconds();

        avgTime += endTime - startTime;
    }

    avgTime /= ITER_NUM;

    std::printf("[BellmanFord cuda]:\t\t[%lf] ms\n", avgTime * 1000);
    std::printf("The minimum distance from %d to %d is: %d\n", srcNode, dstNode, minDist);
    
    return 0;
}