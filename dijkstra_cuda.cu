#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <climits>
#include <fstream>
#include <iostream>
#include "CycleTimer.h"
#define ITER_NUM 10

int minimumDistance(int *dist, bool *visited, int numNodes) {
    int min = INT_MAX, min_index = 0;
    for (int v = 0; v < numNodes; v++) {
        if (!visited[v] && dist[v] < min) {
            min = dist[v];
            min_index = v;
        }
    }
    visited[min_index] = true;
    return min_index;
}

__global__ void dijkstra(int *deviceAdjMatrix, int *deviceDist, bool *deviceVisited, int *deviceMinIndex, int numNodes) {
    int u = deviceMinIndex[0];
    deviceVisited[u] = true;
    int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (!deviceVisited[v] && deviceAdjMatrix[u * numNodes + v] && deviceDist[u] + deviceAdjMatrix[u * numNodes + v] < deviceDist[v])
        deviceDist[v] = deviceDist[u] + deviceAdjMatrix[u * numNodes + v];

}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::printf("usage: ./dijkstra_cuda file.txt srcNode dstNode");
        return 1;
    }

    std::ifstream ifs;
    ifs.open(argv[1], std::ifstream::in);
    if (!ifs.good()) {
        std::printf("[Error] Cannot open file %s\n", argv[1]);
        return 1;
    }    
    int numNodes, numEdges;
    ifs >> numNodes >> numEdges;
    std::printf("[numNodes]: %d [numEdges]: %d\n", numNodes, numEdges);
    int* adjMatrix = (int *)calloc(numNodes * numNodes, sizeof(int));
    
    int srcNode = std::atoi(argv[2]);
    int dstNode = std::atoi(argv[3]);
    std::printf("[srcNode]: %u [dstNode]: %u\n", srcNode, dstNode);

    int source, target, weight;
    for (int i = 0; i < numEdges; i++) {
        ifs >> source >> target >> weight;
        adjMatrix[source * numNodes + target] = weight;
        adjMatrix[target * numNodes + source] = weight;
    }
    std::printf("Successfully construct adjacency matrix\n");
    ifs.close();
    
    hipEvent_t startEvent, endEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&endEvent);

    size_t adjMatrixSize = numNodes * numNodes;
    int *deviceAdjMatrix;
    hipMalloc((void **)&deviceAdjMatrix, adjMatrixSize * sizeof(int));
    hipMemcpy(deviceAdjMatrix, adjMatrix, adjMatrixSize * sizeof(int), hipMemcpyHostToDevice);
    
    int *dist = (int *)calloc(numNodes, sizeof(int));
    int *minIndex = (int *)malloc(1 * sizeof(int));
    bool *visited  = (bool *)calloc(numNodes, sizeof(int));
    for (int i = 0; i < numNodes; i++) {
        dist[i] = INT_MAX;
        visited[i] = false;
    }
    dist[srcNode] = 0;
    minIndex[0] = 0;

    int *deviceDist, *deviceMinIndex; 
    bool *deviceVisited;
    hipMalloc((void **)&deviceDist, numNodes * sizeof(int));
    hipMalloc((void **)&deviceMinIndex, sizeof(int));
    hipMalloc((void **)&deviceVisited, numNodes * sizeof(bool));
    hipMemcpy(deviceDist, dist, numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceVisited, visited, numNodes * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlockRelax(1024, 1);
    dim3 numBlockRelax(numNodes / threadsPerBlockRelax.x, 1);

    hipEventRecord(startEvent);
    for (int i = 0; i < numNodes-1; i++) {
        minIndex[0] = minimumDistance(dist, visited, numNodes);
        hipMemcpy(deviceMinIndex, minIndex, sizeof(int), hipMemcpyHostToDevice);
        dijkstra<<<numBlockRelax, threadsPerBlockRelax>>>(deviceAdjMatrix, deviceDist, deviceVisited, deviceMinIndex, numNodes);
        hipMemcpy(dist, deviceDist, numNodes * sizeof(int), hipMemcpyDeviceToHost);
    }
    hipEventRecord(endEvent);

    hipMemcpy(dist, deviceDist, numNodes * sizeof(int), hipMemcpyDeviceToHost);
    int minDist = dist[dstNode];

    float execTime;
    hipEventElapsedTime(&execTime, startEvent, endEvent);
    std::printf("[Dijkstra cuda]:\t\t[%f] ms\n", execTime);
    std::printf("The minimum distance from %d to %d is: %d\n", srcNode, dstNode, minDist);
    

    hipFree(deviceAdjMatrix);
    hipFree(deviceDist);
    hipFree(deviceMinIndex);
    hipFree(deviceVisited);

    free(adjMatrix);
    free(dist);
    free(minIndex);
    free(visited);
    return 0;
}